#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <mpi.h>
#include <climits>
#include <stdio.h>
#include <string>
#include <algorithm>
using namespace std;

#define max 100000
#define SIZE 1024*1024*4

string inputfile;

void input(char filename[], vector<int> &row, vector<int> &col, vector<int> &data, vector<int> &v, int &dimension){
	string temp;
	int i,j;
	ifstream in;
	in.open(filename);

	if (!in.is_open()){
		cout << "File not found.\n";
	}
	else{
		// Read "Name: Sample_Input" , useless
		in >> temp;
		in >> inputfile;
		
		// Read "Dimension: n"
		in >> temp;
		in >> dimension;
		
		// Read "A"
		in >> temp;
		
		in >> temp;
		char ch11[max];
		int k = 0;
		while (temp.compare("B") != 0){
			i = 0;
			j = 0;
			while (temp[i]!='\0'){
				ch11[j] = temp[i];
				i++;
				j++;
			}
			ch11[j] = '\0';
			if (k==0)
				row.push_back(atoi(ch11));
			if (k==1)
				col.push_back(atoi(ch11));
			if (k==2)
				data.push_back(atoi(ch11));
			k = (k+1)%3;
			in >> temp;
		}
		for (i = 0; i < dimension; i++){
			in >> j;
			v.push_back(j);
		}
	}
}

__global__ void spmv_csr_scalar_kernel(const int num_rows, const int *ptr, const int *indices, const int *data, const int *x, int *y){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < num_rows){
		float dot = 0;
		int row_start = ptr [ row ];
		int row_end = ptr [ row +1];
		for (int j = row_start; j < row_end; j++)
			dot += data[j]*x[indices[j]];
		y[row] += dot ;
	}
}

void sparseMatMul(char filename1[], char filename[]){
	// Initialize the MPI environment
	MPI_Init(NULL, NULL);

	// Get the number of processes
	int totalProcs;
	MPI_Comm_size(MPI_COMM_WORLD, &totalProcs);
	
	// Get the rank of the process
	int procNum;
	MPI_Comm_rank(MPI_COMM_WORLD, &procNum);

	vector<int> r,c,d,v;
	int rBegin = 0;
	int rEnd = 0;
	int dimension;
	vector<int> vec;

	if (procNum == 0){
		vector<int> rows,cols,data1;
		input(filename1,rows,cols,data1,vec,dimension);

		int indicesize = (rows.size())/totalProcs;
		for (int i = 0; i < indicesize; i++){
			r.push_back(rows[i]);
			c.push_back(cols[i]);
			d.push_back(data1[i]);
		}
		for (int i = 1; i < totalProcs; i++){
			if (i == totalProcs-1){
				int toSendSize = int(rows.size()) - (i*indicesize);
				MPI_Send(&toSendSize, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
				if (toSendSize != 0){
					MPI_Send(&rows[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
					MPI_Send(&cols[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
					MPI_Send(&data1[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
				}
			}
			else{
				int toSendSize = indicesize;
				MPI_Send(&toSendSize, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
				if (toSendSize != 0){
					MPI_Send(&rows[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
					MPI_Send(&cols[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
					MPI_Send(&data1[i*indicesize], toSendSize, MPI_INT, i, 0, MPI_COMM_WORLD);
				}

			}
			MPI_Send(&dimension, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
			if (dimension != 0){
				MPI_Send(&vec[0], dimension, MPI_INT, i, 0, MPI_COMM_WORLD);
			}
		}
		rows.clear();
		cols.clear();
		data1.clear();
	}
	else{
		int toReceiveSize = 0;
		MPI_Recv(&toReceiveSize, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		r.resize(toReceiveSize);
		c.resize(toReceiveSize);
		d.resize(toReceiveSize);
		if (toReceiveSize != 0){
			MPI_Recv(&r[0], toReceiveSize, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			MPI_Recv(&c[0], toReceiveSize, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			MPI_Recv(&d[0], toReceiveSize, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		}
		MPI_Recv(&dimension, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		vec.resize(dimension);
		if (dimension != 0){
			MPI_Recv(&vec[0], dimension, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		}
	}
	int indicesize = r.size();
	rBegin = r[0];
	rEnd = r[indicesize-1];
	int ptrsize = rEnd - rBegin + 2;
	
	int* ptr = (int*)malloc(ptrsize*sizeof(int));
	int* indices = (int*)malloc(indicesize*sizeof(int));
	int* data = (int*)malloc(indicesize*sizeof(int));
	int* x = (int*)malloc(dimension*sizeof(int));
	int* y = (int*)malloc((ptrsize-1)*sizeof(int));

	int init = rBegin-1;
	int count = 0;
	int pointer = 0;
	for (int i = 0; i < indicesize; i++){
		indices[i] = c[i];
		data[i] = d[i];
		if (r[i] != init){
			for (int j = 0; j < r[i]-init; j++){
				ptr[pointer] = count;
				pointer ++;
			}
			init = r[i];
		}
		count++;
	}
	ptr[pointer] = count;
	for (int i = 0; i < dimension; i++){
		x[i] = vec[i];
	}
	MPI_Barrier(MPI_COMM_WORLD);
	
	unsigned int threadsPerBlock = 128;
	unsigned int totalBlocks = (SIZE+(threadsPerBlock-1))/threadsPerBlock;

	// if (procNum == 3){
	// 	for (int i = 0; i < ptrsize; i++){
	// 		cout << ptr[i] << " ";
	// 	}
	// 	cout << endl;
	// 	for (int i = 0; i < indicesize; i++){
	// 		cout << indices[i] << " " << data[i] << endl;
	// 	}
	// 	for (int i = 0; i < dimension; i++){
	// 		cout << x[i] << " " ;
	// 	}
	// 	cout << endl;
	// }
	int *cptr, *cindices, *cdata, *cx, *cy;
	hipMalloc((void**)&cptr,ptrsize*sizeof(int));
	hipMalloc((void**)&cindices,indicesize*sizeof(int));
	hipMalloc((void**)&cdata,indicesize*sizeof(int));
	hipMalloc((void**)&cx,dimension*sizeof(int));
	hipMalloc((void**)&cy,(ptrsize-1)*sizeof(int));
	
	hipMemcpy(cptr,ptr,ptrsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cindices,indices,indicesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cdata,data,indicesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cx,x,dimension*sizeof(int),hipMemcpyHostToDevice);
	
	hipMemset(cy,0,(ptrsize-1)*sizeof(int));

	spmv_csr_scalar_kernel<<<totalBlocks, threadsPerBlock>>>(ptrsize-1,cptr,cindices,cdata,cx,cy);

	hipMemcpy(y,cy,(ptrsize-1)*sizeof(int),hipMemcpyDeviceToHost);

	///////////// Cuda se merge the vectors ////////////////////
	// int* y = (int*)malloc((ptrsize-1) * sizeof(int));
	if (procNum == 3)
		for (int i = 0; i < ptrsize-1; i++){
			cout << y[i] << " ";
		}
	if (procNum != 0){
		MPI_Send(&rBegin, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
		MPI_Send(&rEnd, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
		if ((rEnd - rBegin + 1) != 0)
			MPI_Send(&y[0], (rEnd - rBegin + 1), MPI_INT, 0, 0, MPI_COMM_WORLD);
	}
	else{
		ofstream out(filename, ios::out);
		int j = 0;
		if (rBegin != 0){
			for (; j < rBegin; j++){
				out << 0 << endl;
			}
		}
		for (; j < rEnd; j++){
			out << y[j] << endl;
		}
		int last = y[rEnd];
		int size1,size2;
		for (int i = 1; i < totalProcs; i++){
			MPI_Recv(&size1, 1, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			MPI_Recv(&size2, 1, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			vector<int> v;
			v.resize(size2 - size1 + 1);
			if ((size2 - size1 + 1) != 0)
				MPI_Recv(&v[0], size2-size1+1, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			if (size1 == rEnd){
				if (size1 == size2)
					last += v[0];
				else{
					out << v[0]+last << endl;
					for (j = 1; j < v.size()-1; j++){
						out << v[j] << endl;
					}
					last = v[v.size()-1];
					rEnd = size2;
				}
			}
			else{
				out << last << endl;
				for (j = rEnd+1; j < size1; j++){
					out << 0 << endl;
				}
				if (size1 == size2){
					last = v[0];
					rEnd = size2;
				}
				else{
					out << v[0] << endl;
					for (j = 1; j < v.size()-2; j++){
						out << v[j] << endl;
					}
					last = v[v.size()-1];
					rEnd = size2;
				}
			}		
		}
		out << last << endl;
		for (int i = rEnd+1; i < dimension; i++){
			out << 0 << endl;
		}
		out.close();
	}
	// Finalize the MPI environment.
	MPI_Finalize();
}

int main(int argc, char** argv){
	sparseMatMul(argv[1], argv[2]);
}
